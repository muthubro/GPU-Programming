#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "stb_image/stb_image.h"

struct vec2
{
    double x, y;
};

vec2 operator +(vec2 a, vec2 b)
{
    return vec2(a.x + b.x, a.y + b.y);
}

vec2 operator *(vec2 a, double s)
{
    return vec2(a.x * s, a.y * s);
}

struct vec3
{
    double x, y, z;

    vec3(double val) { x = val; y = val; z = val; }
    vec2 xy() { return vec2(x, y); }
};

vec3 operator +(vec3 a, vec3 b)
{
    return vec3(a.x + b.x, a.y + b.y, a.z + b.z);
}

vec3 operator -(vec3 a, vec3 b)
{
    return vec3(a.x - b.x, a.y - b.y, a.z - b.z);
}

vec3 pow(vec3 a, vec3 p)
{
    return vec3(pow(a.x, p.x), pow(a.y, p.y), pow(a.z, p.z));
}

#define EXP 2.71828

vec3 exp(vec3 vec)
{
    return pow(vec3(EXP), vec);
}

double dot(vec3 a, vec3 b)
{
    return a.x*b.x + a.y*b.y + a.z*b.z;
}

vec3 mix(vec3 a, vec3 b, double r)
{
    return a + (b - a) * r;
}

double clamp(double a, double l, double h)
{
    return a < l ? l : (a > h ? h : a);
}

struct Image
{
    float* data;
    int width;
    int height;
}

vec3 texture(Image data, vec2 coords)
{
    int pix = coord.y * image.width + coord.x;
    return vec3(data[pix*3], data[pix*3+1], data[pix*3+2]);
}

const float PI = 3.14159265359;

// Trowbridge-Reitz GGX Normal Distribution Function
float Distribution(vec3 N, vec3 H, float roughness)
{
	float a = roughness * roughness;
	float a2 = a * a;
	float NdotH = max(dot(N, H), 0.0);
	float NdotH2 = clamp(NdotH * NdotH, 0.0, 1.0);

	float num = a2;
	float denom = NdotH2 * (a2 - 1.0) + 1.0;
	denom = PI * denom * denom;

	return num / max(denom, 0.0000001);
}

// Schlick GGX Geometry
float GeometrySchlickGGX(float NdotV, float roughness)
{
	float r = roughness + 1.0;
	float k = r * r / 8.0;

	float num = NdotV;
	float denum = NdotV * (1.0 - k) + k;

	return num / denum;
}

// Smith's Method
float Geometry(vec3 N, vec3 L, vec3 V, float roughness)
{
	float NdotL = max(dot(N, L), 0.0);
	float NdotV = max(dot(N, V), 0.0);
	float ggx1 = GeometrySchlickGGX(NdotL, roughness);
	float ggx2 = GeometrySchlickGGX(NdotV, roughness);

	return ggx1 * ggx2;
}

// Fresnel-Schlick approximation
vec3 Fresnel(float cosTheta, vec3 F0, float roughness)
{
	return F0 + (max(vec3(1.0 - roughness), F0) - F0) * pow(1.0 - cosTheta, 5.0);
}

vec2 ParallaxCalculation(vec2 texCoord, vec3 viewDir)
{
	const float minLayers = 8;
	const float maxLayers = 32;
	float numLayers = mix(maxLayers, minLayers, max(dot(vec3(0.0, 0.0, 1.0), viewDir), 0.0));
	float layerDepth = 1.0 / numLayers;

	float currentLayerDepth = 0.0;
	vec2 P = viewDir.xy() / viewDir.z * 0.03;
	vec2 deltaTexCoords = P / numLayers;

	vec2 currentTexCoords = texCoord;
	float currentDepthMapValue = texture(u_HeightMap, texCoord).x;

	while (currentLayerDepth < currentDepthMapValue)
	{
		currentTexCoords -= deltaTexCoords;
		currentDepthMapValue = 1.0 - texture(u_HeightMap, currentTexCoords).x;
		currentLayerDepth += layerDepth;
	}

	vec2 prevTexCoords = currentTexCoords + deltaTexCoords;

	float afterDepth = currentDepthMapValue - currentLayerDepth;
	float beforeDepth = texture(u_HeightMap, prevTexCoords).x - currentLayerDepth + layerDepth;

	float weight = afterDepth / (afterDepth - beforeDepth);
	vec2 finalTexCoords = prevTexCoords * weight + currentTexCoords * (1.0 - weight);

	return finalTexCoords;
}

double Length(vec3 vec)
{
    return sqrt(vec.x*vec.x + vec.y*vec.y + vec.z*vec.z);
}

vec3 Normalize(vec3 vec)
{
    vec *= 1.0 / Length(vec);
    return vec;
}

// Cook-Torrance BRDF
__global__ void PBR(Image& img, vec3 viewPos, vec3 worldPos, vec2 texCoord, 
    Image normalMap, vec3 lightColors[4], vec3 lightPositions[4], bool ibl, 
    Image prefliterMap, Image BRDFLUT, double exposure)
{
    vec3 V = Normalize(viewPos - worldPos);

    vec2 texCoords = ParallaxCalculation(texCoord, V);
    vec3 N = texture(normalMap, texCoord) * 2.0 - vec3(1.0, 1.0, 1.0);

    vec3 albedo;
	float metallic;
	float roughness;
	float ao;

    albedo = pow(texture(u_AlbedoMap, texCoords), vec3(2.2));
    metallic = texture(u_MetallicMap, texCoords).x;
    roughness = texture(u_RoughnessMap, texCoords).x;
    ao = texture(u_AOMap, texCoords).x;

    vec3 F0 = vec3(0.04);
	F0 = mix(F0, albedo, metallic);
	vec3 Lo = vec3(0.0);
	for (int i = 0; i < 4; i++)
	{
		vec3 L = Normalize(lightPositions[i] - worldPos);
		vec3 H = Normalize(L + V);

		float distance = Length(lightPositions[i] - worldPos);
		float attenuation = 1.0 / (distance * distance);
		vec3 radiance = lightColors[i] * attenuation;

		// Cook-Torrance BRDF
		float NDF = Distribution(N, H, roughness);
		float G = Geometry(N, L, V, roughness);
		vec3 F = Fresnel(clamp(dot(H, V), 0.0, 1.0), F0, roughness);

		vec3 num = NDF * G * F;
		float denom = 4 * max(dot(N, V), 0.0) * max(dot(N, L), 0.0);
		vec3 specular = num / max(denom, 0.001);

		vec3 k_d = vec3(1.0) - F;
		k_d *= 1.0 - metallic;
		vec3 diffuse = k_d * albedo / PI;

		vec3 BRDF = diffuse + specular;
		float NdotL = max(dot(N, L), 0.0);

		Lo += BRDF * radiance * NdotL; 
	}

	vec3 ambient;
	if (ibl)
	{
		// IBL
		vec3 k_s = Fresnel(clamp(dot(N, V), 0.0, 1.0), F0, roughness);
		vec3 k_d = 1.0 - k_s;
		vec3 irradiance = texture(u_IrradianceMap, N);
		vec3 diffuse = irradiance * albedo;

		const float MAX_REFLECTION_LOD = 4.0;
		vec3 prefilteredColor = texture(prefliterMap, R);
		vec2 BRDF = texture(BRDFLUT, vec2(max(dot(N, V), 0.0), roughness)).xy;
		vec3 specular = prefilteredColor * (F0 * BRDF.x + BRDF.y);

		ambient = (k_d * diffuse + specular) * ao;
	}
	else
	{
		ambient = vec3(0.03) * albedo * ao;
	}
	
	vec3 color = ambient + Lo;

	// Tone mapping
	color = vec3(1.0) - exp(-color * exposure);
	// Gamma correction
	color = pow(color, vec3(1.0 / 2.2));

	img[blockIdx.x * img.height + blockIdx.y] = color;
}

int Lighting(int* argc, char** argv)
{
    Image img, normalMap, prefilterMap, BRDFLUT;
    bool ibl = atoi(argv[0]);
    double exposure = atod(argv[1]);

    int channels;
    img.data = stbi_loadf("pirate-gold-bl/pirate-gold_albedo.png", &img.width, &img.height, &channels, 3);
    normalMap.data = stbi_loadf("pirate-gold-bl/pirate-gold_normal-ogl.png", &normalMap.width, &normalMap.height, &channels, 0);
    prefilterMap.data = stbi_loadf("Newport_Loft/Newport_Loft_8k.png", &prefilterMap.width, &prefilterMap.height, &channels, 0);
    BRDFLUT.data = stbi_loadf("BRDF_LUT.tga", &img.width, &img.height, &channels, 0);

    vec3 lightPositions[] = {
        vec3(-10.0f,  10.0f, 10.0f),
        vec3(10.0f,  10.0f, 10.0f),
        vec3(-10.0f, -10.0f, 10.0f),
        vec3(10.0f, -10.0f, 10.0f),
    };
    vec3 lightColors[] = {
        vec3(1000.0f, 1000.0f, 1000.0f),
        vec3(300.0f, 300.0f, 300.0f),
        vec3(300.0f, 300.0f, 300.0f),
        vec3(300.0f, 300.0f, 300.0f)
    };

    PBR<<<vec3(img.width, image.height, 1), 1>>>(img, vec3(0.0, 0.0, 0.0), vec3(0.0, 0.0, 0.0), vec2(0.2, 0.5), 
        img, lightColors, lightPositions, ibl, prefilterMap, BRDFLUT, exposure);

    output = fopen("assets/textures/lighting.png", "wb");
    fprintf(output, (char*)img.data);
    fclose(output);

    return 0;
}